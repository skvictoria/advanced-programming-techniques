#include <stdio.h> 
#include <iostream>
// http://developer.download.nvidia.com/compute/cuda/4_0/toolkit/docs/online/group__CUDART__DEVICE_g5aa4f47938af8276f08074d09b7d520c.html
using namespace std;

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Integrated Device: %s\n", prop.integrated?"True":"False");

    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

	cout << "  Num. Multiprocessors: " << prop.multiProcessorCount << endl;
	
	cout << "  Max. Threads per block: " << prop.maxThreadsPerBlock << endl;
	// Maximum size of each dimension of a grid.
	cout << "  Max Grid Size: (" 
	     << prop.maxGridSize[0] << ", "
		 << prop.maxGridSize[1] << ", "
		 << prop.maxGridSize[2] << ")" << endl;
	cout << "  Max Block Size: (" 
	     << prop.maxThreadsDim[0] << ", "
		 << prop.maxThreadsDim[1] << ", "
		 << prop.maxThreadsDim[2] << ")" << endl;
		 
	cout << endl;
		 
  }
}