/*
Author: Seulgi Kim
Class: ECE6122 A
Last Date Modified: 11/07/2023

Description:

This code gets number of walkers and number of each steps that walkers take,
simulate a random walk (either east, west, south, north) of each walker, and calculate the average distance.

*/
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <numeric>

unsigned long long num_steps = 10000;
unsigned long long num_walkers = 1000;

/**
 * @brief This method extract the valid natural number from the input stream from the user.
 * @param strInputStream The incoming string stream
 * @param validNum return extracted valid natural number.
 * @return 1 if succeed extracting, 0 if not.
*/
uint8_t ExtractValidNaturalNum(const char* strInputStream, unsigned long long &validNum)
{
    uint8_t u8Success = 1U;
    char cSubString;
    std::string strresult(strInputStream);

    if (strresult.length() > 0U)
    {   
        u8Success = 1U;
        // for each letter in a number,
        for (uint8_t u8LetterIdx = 0U; u8LetterIdx < strresult.length(); u8LetterIdx++)
        {
            cSubString = (strresult.substr(u8LetterIdx, 1))[0];
            // if any one of the input is not digit, failed.
            if (isdigit(cSubString) == 0)
            {
                u8Success = 0U;
                break;
            }
        }

        if (u8Success == 1U)
        {
            validNum = strtoull(strInputStream, nullptr, 10);
        }
    }
    else
    {
        u8Success = 0U;
    }

    return u8Success;
}

/**
 * @brief This method calculate the distance for walkers.
 * @param state random state.
 * @param distance distance for walker.
 * @param numWalkers number of walkers from the user.
 * @param numSteps number of steps from the user.
 * @param seed seed for generating random state.
*/
__global__ void calculateDistanceperWalker(hiprandState* state, float* distance, unsigned long long numWalkers
                                        , unsigned long long numSteps, int seed) 
{
    unsigned long long tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numWalkers)
    {
        return;
    }
    float randVal;
    unsigned long long x = 0, y = 0;
    hiprand_init(static_cast<unsigned int>(seed), tid, 0, &state[tid]);

    // #1 Optimization: reduce if-condition for random walk (either east, west, south, north)
        // east  if 0.0f  < randVal < 0.25f
        // west  if 0.25f < randVal < 0.5f
        // south if 0.5f  < randVal < 0.75f
        // north if 0.75f < randVal < 1.0f
    for (unsigned long long step = 0; step < numSteps; ++step)
    {
        randVal = hiprand_uniform(&state[tid]);
        x += (randVal < 0.25f) - (randVal >= 0.25f && randVal < 0.5f);
        y += (randVal >= 0.75f) - (randVal >= 0.5f && randVal < 0.75f);
    }
    distance[tid] = sqrtf(x * x + y * y);
}

/**
 * @brief This method calculate the distance for walkers.
 * @param walkers walkers array for host memory
 * @param d_walkers walkers array for device memory
 * @param d_states states array (for generating random states) for device memory
*/
void NormalCudaMalloc(float* walkers, float* d_walkers, hiprandState* d_states)
{
    // Initialize the value.
    float averageDistance = 0.0f;
    float microsec;
    hipEvent_t startEvent, stopEvent; 
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // Start Timing.
    hipEventRecord(startEvent, 0);

    // Allocate host memory.
    walkers = (float*)malloc(sizeof(float) * num_walkers);

    // Allocate device memory.
    hipMalloc((void**)&d_walkers, sizeof(float) * num_walkers);
    hipMalloc((void**)&d_states, sizeof(hiprandState) * num_walkers);

    calculateDistanceperWalker<<<(num_walkers+256)/256, 256, 0, stream>>>(d_states, d_walkers, num_walkers, num_steps, time(NULL));

    // Transfer data back to host memory.
    // #2 Optimization: Use Cuda Async operation - overlap of kernel execution & Data Transfer.
    hipMemcpyAsync(walkers, d_walkers, sizeof(float) * num_walkers, hipMemcpyDeviceToHost, stream);

    // Make sure block CPU thread before finializing the cuda async processes.
    hipStreamSynchronize(stream);
    averageDistance = std::accumulate(walkers, walkers + num_walkers, 0.0f) / num_walkers;

    // Deallocate device memory.
    hipFree(d_walkers);
    hipFree(d_states);

    // Deallocate host memory.
    free(walkers);

    // Destroy Stream.
    hipStreamDestroy(stream);

    // Finish Timing.
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&microsec, startEvent, stopEvent);
    std::cout << "Normal CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << microsec * 1000 << std::endl;
    std::cout << "    Average distance from origin: " << averageDistance << std::endl;
}

/**
 * @brief This method calculate the distance for walkers.
 * @param walkers walkers array for host memory
 * @param d_walkers walkers array for device memory
 * @param d_states states array (for generating random states) for device memory
*/
void PinnedCudaMalloc(float** walkers, float* d_walkers, hiprandState* d_states)
{
    // Initialize the value.
    float averageDistance = 0.0f;
    float microsec;
    hipEvent_t startEvent, stopEvent;
    hipStream_t stream;
    hipStreamCreate(&stream);
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // Start Timing.
    hipEventRecord(startEvent, 0);

    // Allocate host memory.
    hipHostMalloc((void**)walkers, sizeof(float) * num_walkers, hipHostMallocDefault);

    // Allocate device memory.
    hipMalloc((void**)&d_walkers, sizeof(float) * num_walkers);
    hipMalloc((void**)&d_states, sizeof(hiprandState) * num_walkers);

    calculateDistanceperWalker<<<(num_walkers+256)/256, 256, 0, stream>>>(d_states, d_walkers, num_walkers, num_steps, time(NULL));

    // Transfer data back to host memory.
    // #3 Optimization: Use Cuda Async operation - overlap of kernel execution & Data Transfer.
    hipMemcpyAsync(*walkers, d_walkers, sizeof(float) * num_walkers, hipMemcpyDeviceToHost, stream);
    
    // Make sure block CPU thread before finializing the cuda async processes.
    hipStreamSynchronize(stream);
    averageDistance = std::accumulate(*walkers, *walkers + num_walkers, 0.0f) / num_walkers;

    // Deallocate device memory.
    hipFree(d_walkers);
    hipFree(d_states);

    // Deallocate host memory.
    hipHostFree(*walkers);

    // Destroy Stream.
    hipStreamDestroy(stream);

    // Finish Timing.
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&microsec, startEvent, stopEvent);
    std::cout << "Pinned CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << microsec * 1000 << std::endl;
    std::cout << "    Average distance from origin: " << averageDistance << std::endl;
}

/**
 * @brief This method calculate the distance for walkers.
 * @param d_walkers walkers array for device memory.
 * @param d_states states array (for generating random states) for device memory
*/
void ManagedCudaMalloc(float* d_walkers, hiprandState* d_states)
{
    // Initialize the value.
    float averageDistance = 0.0f;
    float microsec;
    hipEvent_t startEvent, stopEvent; 
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    // Start Timing.
    hipEventRecord(startEvent, 0);

    // Allocate memory.
    hipMallocManaged(&d_walkers, sizeof(float) * num_walkers);
    hipMallocManaged(&d_states, sizeof(hiprandState) * num_walkers);

    calculateDistanceperWalker<<<(num_walkers+256)/256, 256>>>(d_states, d_walkers, num_walkers, num_steps, time(NULL));
    hipDeviceSynchronize();
    averageDistance = std::accumulate(d_walkers, d_walkers + num_walkers, 0.0f) / num_walkers;

    // Deallocate device memory.
    hipFree(d_walkers);
    hipFree(d_states);

    // Finish Timing.
    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);
    hipEventElapsedTime(&microsec, startEvent, stopEvent);
    std::cout << "Managed CUDA memory Allocation:" << std::endl;
    std::cout << "    Time to calculate(microsec): " << microsec * 1000 << std::endl;
    std::cout << "    Average distance from origin: " << averageDistance << std::endl;
}

int main(int argc, char* argv[])
{
    // Initialize the variables.
    float *walkers = 0;
    float *d_walkers = 0;
    hiprandState *d_states = 0;

    // Get the input from user.
    for (int i = 1; i < argc - 1; ++i)
    {
        if (strcmp(argv[i], "-W") == 0)
        {
            if (ExtractValidNaturalNum(argv[++i], num_walkers) == 0U)
            {
                std::cout << "Invalid num_walkers" << std::endl;
                return EXIT_FAILURE;
            }
        }
        if (strcmp(argv[i], "-I") == 0)
        {
            if (ExtractValidNaturalNum(argv[++i], num_steps) == 0U)
            {
                std::cout << "Invalid num_steps" << std::endl;
                return EXIT_FAILURE;
            }
        }
    }
    NormalCudaMalloc(walkers, d_walkers, d_states);
    
    PinnedCudaMalloc(&walkers, d_walkers, d_states);
    
    ManagedCudaMalloc(d_walkers, d_states);

    std::cout << "Bye" << std::endl;

    return EXIT_SUCCESS;
}